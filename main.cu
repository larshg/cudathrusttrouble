#include "hip/hip_runtime.h"
#include "data_source.hpp"
#include <iostream>
#include <cstdlib>
#include <hip/hip_runtime_api.h>
#include <thrust/reduce.h>
#include <thrust/device_ptr.h>
#include <thrust/device_vector.h>

template <typename T>
struct LARGERT
{
  __host__ __device__ __forceinline__
    T operator()(const T& x, const T& y) const {
    float4 result;
    printf("Adress first: %p\n", (void*)&x);
    printf("Adress second: %p\n", (void*)&y);
    printf("comparing LargerT: %f, %f, %f with %f %f %f\n", x.x, x.y, x.z, y.x, y.y, y.z);
    result.x = fmax(x.x, y.x);
    result.y = fmax(x.y, y.y);
    result.z = fmax(x.z, y.z);
    return result;
  }
};

template <typename T>
struct LESST
{
  __host__ __device__ __forceinline__
    T operator()(const T& x, const T& y) const {
    float4 result;
    printf("Adress first: %p\n", (void*)&x);
    printf("Adress second: %p\n", (void*)&y);
    printf("comparing LessT: %f, %f, %f with %f %f %f\n", x.x, x.y, x.z, y.x, y.y, y.z);
    result.x = fmin(x.x, y.x);
    result.y = fmin(x.y, y.y);
    result.z = fmin(x.z, y.z);
    return result;
  }
};

void error(const char* error_string, const char* file, const int line, const char* func)
{
  std::cout << "Error: " << error_string << "\t" << file << ":" << line << std::endl;
  exit(EXIT_FAILURE);
}

static inline void ___cudaSafeCall(hipError_t err, const char* file, const int line, const char* func = "")
{
  if (hipSuccess != err)
    error(hipGetErrorString(err), file, line, func);
}

#define cudaSafeCall(expr)  ___cudaSafeCall(expr, __FILE__, __LINE__)


void FilterPoints(float4* baseAddress, size_t sizeBytes_)
{
  /** \brief Device pointer. */
  void* data_;

  cudaSafeCall(hipMalloc(&data_, sizeBytes_));
  cudaSafeCall(hipMemcpy(data_, baseAddress, sizeBytes_, hipMemcpyHostToDevice));
  cudaSafeCall(hipDeviceSynchronize());


  float4 max;
  max.x = max.y = max.z = FLT_MAX;
  max.w = 0;

  float4 min;
  min.x = min.y = min.z = -FLT_MAX;
  min.w = 0;

  thrust::device_ptr<float4> beg((float4*)data_);
  thrust::device_ptr<float4> end = beg + sizeBytes_ / 16;
  std::cout << "Before reduce" << std::endl;
  float4 minp = thrust::reduce(beg, end, max, LESST<float4>{});
  float4 maxp = thrust::reduce(beg, end, min, LARGERT<float4>{});

  std::cout << "minp is: " << minp.x << "," << minp.y << "," << minp.z << std::endl;
  std::cout << "maxp is: " << maxp.x << "," << maxp.y << "," << maxp.z << std::endl;
}

int main(void)
{
  DataGenerator data;
  data.data_size = 2;
  data.tests_num = 10000;
  data.cube_size = 1024.f;
  data.max_radius = data.cube_size / 30.f;
  data.shared_radius = data.cube_size / 30.f;
  data.printParams();
  //generate
  data();

  size_t sizeBytes_ = data.data_size * 16;

  std::cout << "Before filter" << std::endl;
  FilterPoints(&data.points[0], sizeBytes_);

  return 0;
}